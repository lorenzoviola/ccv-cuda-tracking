#include "hip/hip_runtime.h"

float elapsedtime;

__global__ void convert(int width, int height, uchar4 *gpu_in)
{
	
	int tx = threadIdx.x + (blockIdx.x * blockDim.x);
	int ty = threadIdx.y + (blockIdx.y * blockDim.y);
	int offset = tx + ty * blockDim.x*gridDim.x;

	if(offset < width * height)
	{	
		float color = 0.3 * (gpu_in[offset].x) + 0.6 * (gpu_in[offset].y) + 0.1 * (gpu_in[offset].z);
		gpu_in[offset].x = color;
		gpu_in[offset].y = color;
		gpu_in[offset].z = color;
		gpu_in[offset].w = 0;
	}	
	
}
///////////////// CUDA function call wrapper /////////////////
void gpu_grayscale(int width, int height, unsigned char *in)
{
	uchar4 *gpu_in;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	////////////////////////// Time consuming Task //////////////////////////////////	
	hipMalloc((void **)&gpu_in, (width * height * 4 * sizeof(unsigned char)));
	hipMemcpy(gpu_in, in, (width * height * 4 * sizeof(unsigned char)), hipMemcpyHostToDevice);

	dim3 grid(18,18);
	dim3 block(16,16);
	convert<<<grid,block>>>(width, height, gpu_in);

	hipMemcpy( in, gpu_in, (width * height * 4 * sizeof(unsigned char)), hipMemcpyDeviceToHost);
	/////////////////////////////////////////////////////////////////////////////////

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
}

